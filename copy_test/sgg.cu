#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

const int kWarmUpTurns = 10;
const size_t kWarmUpSize = ((size_t) 2) * 1024 * 1024 * 1024; // 2GB

float DirectCopyTest(int device_id0, int device_id1, size_t size) { int *pointers[2];

    hipSetDevice(device_id0);
    // cudaDeviceEnablePeerAccess(device_id1, 0);
    hipMalloc(&pointers[0], size);

    hipSetDevice(device_id1);
    // cudaDeviceEnablePeerAccess(device_id0, 0);
    hipMalloc(&pointers[1], size);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // Check available
    // int avail;
    // cudaDeviceCanAccessPeer(&avail, device_id0, device_id1);
    // printf("[%d]", avail);

    // Warm Up
    int index;
    for (index = 0; index < kWarmUpTurns; ++index) {
        hipMemcpy(pointers[0], pointers[1], kWarmUpSize, hipMemcpyDeviceToDevice);
    }

    hipEventRecord(begin);
    hipMemcpy(pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, begin, end);
    elapsed /= 1000;

    hipSetDevice(device_id0);
    // cudaDeviceDisablePeerAccess(device_id1);
    hipFree(pointers[0]);

    hipSetDevice(device_id1);
    // cudaDeviceDisablePeerAccess(device_id0);
    hipFree(pointers[1]);

    hipEventDestroy(end);
    hipEventDestroy(begin);

    return elapsed;
}


float P2PCopyTest(int device_id0, int device_id1, size_t size) {
    int *pointer0;
    int *pointer1;

    hipSetDevice(device_id0);
    hipMalloc(&pointer0, size);

    hipSetDevice(device_id1);
    hipMalloc(&pointer1, size);
    
    // Set device back to device0
    hipSetDevice(device_id0);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // Check available
    // int avail;
    // cudaDeviceCanAccessPeer(&avail, device_id0, device_id1);
    // printf("[%d]", avail);

    // Warm Up
    int index;
    for (index = 0; index < kWarmUpTurns; ++index) {
        // cudaMemcpyPeer(pointer0, device_id0, pointer1, device_id1, kWarmUpSize);
        hipMemcpyPeer(pointer1, device_id1, pointer0, device_id0, kWarmUpSize);
        // cudaMemcpyPeer(pointer1, pointer0, kWarmUpSize, cudaMemcpyDeviceToDevice);
    }

    hipEventRecord(begin);
    // cudaEventSynchronize(begin);
    hipMemcpyPeer(pointer0, device_id0, pointer1, device_id1, size);
    hipDeviceSynchronize();
    // cudaMemcpy(pointers[1], pointers[0], size, cudaMemcpyDeviceToDevice);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, begin, end);

    hipSetDevice(device_id0);
    hipFree(pointer0);

    hipSetDevice(device_id1);
    hipFree(pointer1);

    hipEventDestroy(end);
    hipEventDestroy(begin);

    return elapsed;
}

float PinCopyTest(int device_id0, int device_id1, size_t size) {
    int *pointer_device0;
    int *pointer_device1;
    int *pointer_host;

    hipHostAlloc((void**)&pointer_host, size, hipHostMallocDefault);
    hipSetDevice(device_id0);
    hipMalloc((void**)&pointer_device0, size);
    hipSetDevice(device_id1);
    hipMalloc((void**)&pointer_device1, size);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // Warm Up
    hipEventRecord(begin);
    // cudaEventSynchronize(begin);
    hipMemcpy(pointer_host, pointer_device0, size, hipMemcpyDeviceToHost);
    hipMemcpy(pointer_device1, pointer_host, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    // cudaMemcpy(pointers[1], pointers[0], size, cudaMemcpyDeviceToDevice);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, begin, end);

    hipHostFree(pointer_host);

    hipSetDevice(device_id0);
    hipFree(pointer_device0);
    hipSetDevice(device_id1);
    hipFree(pointer_device1);

    hipEventDestroy(end);
    hipEventDestroy(begin);

    return elapsed;
}

int main() {
    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    if (numGPUs < 2) {
        std::cout << "Error: At least two GPUs are required." << std::endl;
        return 0;
    }
    

    printf("Bandwidth test result:\n");

    for (int index1 = 0; index1 < numGPUs; index1++) {
        hipSetDevice(index1);
        for (int index2 = 0; index2 < numGPUs; index2++) {
            hipDeviceEnablePeerAccess(index2, 0);
        }
    }
    
    for (int index1 = 0; index1 < numGPUs; index1++) {
        for (int index2 = 0; index2 < numGPUs; index2++) {
            size_t data_size = 128 * 1024 * 1024 * sizeof(int);
            float time = P2PCopyTest(index1, index2, data_size);
            printf("%10.2f", time);
        }
        printf("\n");
    }

    printf("\n");
    printf("Pin memory copy test:\n");
    for (int index1 = 0; index1 < numGPUs; index1++) {
        for (int index2 = 0; index2 < numGPUs; index2++) {
            size_t data_size = 128 * 1024 * 1024 * sizeof(int);
            float time = PinCopyTest(index1, index2, data_size);
            printf("%10.2f", time);
        }
        printf("\n");
    }

    return 0;
}
