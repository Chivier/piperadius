#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

const int kWarmUpTurns = 10;
const size_t kWarmUpSize = 8 * 1024 * 1024; // 8 MB

float DirectCopyTest(int device_id0, int device_id1, size_t size) {
    int *pointers[2];

    hipSetDevice(device_id0);
    // cudaDeviceEnablePeerAccess(device_id1, 0);
    hipMalloc(&pointers[0], size);

    hipSetDevice(device_id1);
    // cudaDeviceEnablePeerAccess(device_id0, 0);
    hipMalloc(&pointers[1], size);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // Check available
    // int avail;
    // cudaDeviceCanAccessPeer(&avail, device_id0, device_id1);
    // printf("[%d]", avail);

    // Warm Up
    int index;
    for (index = 0; index < kWarmUpTurns; ++index) {
        hipMemcpy(pointers[0], pointers[1], kWarmUpSize, hipMemcpyDeviceToDevice);
    }

    hipEventRecord(begin);
    hipMemcpy(pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, begin, end);
    elapsed /= 1000;

    hipSetDevice(device_id0);
    // cudaDeviceDisablePeerAccess(device_id1);
    hipFree(pointers[0]);

    hipSetDevice(device_id1);
    // cudaDeviceDisablePeerAccess(device_id0);
    hipFree(pointers[1]);

    hipEventDestroy(end);
    hipEventDestroy(begin);

    return elapsed;
}


float P2PCopyTest(int device_id0, int device_id1, size_t size) {
    int *pointer0;
    int *pointer1;

    hipSetDevice(device_id0);
    hipMalloc(&pointer0, size);

    hipSetDevice(device_id1);
    hipMalloc(&pointer1, size);
    
    // Set device back to device0
    hipSetDevice(device_id0);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // Check available
    // int avail;
    // cudaDeviceCanAccessPeer(&avail, device_id0, device_id1);
    // printf("[%d]", avail);

    // Warm Up
    int index;
    for (index = 0; index < kWarmUpTurns; ++index) {
        // cudaMemcpyPeer(pointer0, device_id0, pointer1, device_id1, kWarmUpSize);
        hipMemcpyPeer(pointer1, device_id1, pointer0, device_id0, kWarmUpSize);
        // cudaMemcpyPeer(pointer1, pointer0, kWarmUpSize, cudaMemcpyDeviceToDevice);
    }

    hipEventRecord(begin);
    // cudaEventSynchronize(begin);
    hipMemcpyPeer(pointer0, device_id0, pointer1, device_id1, size);
    hipDeviceSynchronize();
    // cudaMemcpy(pointers[1], pointers[0], size, cudaMemcpyDeviceToDevice);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, begin, end);
    elapsed /= 1000;

    hipSetDevice(device_id0);
    hipFree(pointer0);

    hipSetDevice(device_id1);
    hipFree(pointer1);

    hipEventDestroy(end);
    hipEventDestroy(begin);

    return elapsed;
}

float PinCopyTest(int device_id, size_t size) {
    int *pointer_device;
    int *pointer_host;

    hipHostAlloc((void**)&pointer_host, size, hipHostMallocDefault);
    hipMalloc((void**)&pointer_device, size);
    memset(pointer_host, 1, size);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // Warm Up
    int index;
    for (index = 0; index < kWarmUpTurns; ++index) {
        hipMemcpy(pointer_device, pointer_host, kWarmUpSize, hipMemcpyHostToDevice);
    }

    hipEventRecord(begin);
    // cudaEventSynchronize(begin);
    hipMemcpy(pointer_device, pointer_host, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    // cudaMemcpy(pointers[1], pointers[0], size, cudaMemcpyDeviceToDevice);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, begin, end);
    elapsed /= 1000;

    hipHostFree(pointer_host);
    hipFree(pointer_device);

    hipEventDestroy(end);
    hipEventDestroy(begin);

    return elapsed;
}

int main() {
    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    if (numGPUs < 2) {
        std::cout << "Error: At least two GPUs are required." << std::endl;
        return 0;
    }
    

    printf("Bandwidth test result:\n");

    for (int index1 = 0; index1 < numGPUs; index1++) {
        hipSetDevice(index1);
        for (int index2 = 0; index2 < numGPUs; index2++) {
            hipDeviceEnablePeerAccess(index2, 0);
        }
    }
    
    for (int index1 = 0; index1 < numGPUs; index1++) {
        for (int index2 = 0; index2 < numGPUs; index2++) {
            size_t data_size = 128 * 1024 * 1024 * sizeof(int);
            float time = P2PCopyTest(index1, index2, data_size);
            float bandwidth = (data_size / 1024 / 1024 / 1024.0) / (time); // GB/s
            printf("%10.2f", bandwidth);
        }
        printf("\n");
    }

    printf("\n");
    printf("Pin memory copy test:\n");
    for (int index = 0; index < numGPUs; ++index) {
        size_t data_size = 128 * 1024 * 1024 * sizeof(int);
        float time = PinCopyTest(index, data_size);
        float bandwidth = (data_size / 1024 / 1024 / 1024.0) / (time); // GB/s
        printf("%10.2f", bandwidth);
    }

    return 0;
}
