#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

float P2PCopyTest(int device_id1, int device_id2, size_t size) {
    int *pointers[2];

    hipSetDevice(device_id1);
    hipDeviceEnablePeerAccess(device_id2, 0);
    hipMalloc(&pointers[0], size);

    hipSetDevice(device_id2);
    hipDeviceEnablePeerAccess(device_id1, 0);
    hipMalloc(&pointers[1], size);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    hipEventRecord(begin);
    hipMemcpyAsync(pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, begin, end);
    elapsed /= 1000;

    hipSetDevice(device_id1);
    hipFree(pointers[0]);

    hipSetDevice(device_id2);
    hipFree(pointers[1]);

    hipEventDestroy(end);
    hipEventDestroy(begin);

    return elapsed;
}

int main() {
    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    if (numGPUs < 2) {
        std::cout << "Error: At least two GPUs are required." << std::endl;
        return 0;
    }

    for (int index1 = 0; index1 < numGPUs; index1++) {
        for (int index2 = 0; index2 < numGPUs; index2++) {
            size_t data_size = 64 * 1024 * 1024;
            float time = P2PCopyTest(index1, index2, data_size);
            float bandwidth = (data_size / 1024 / 1024 / 1024.0) / (time); // GB/s
            printf("%f,", bandwidth * bandwidth);
        }
        printf("\n");
    }

    return 0;
}
